
#include <hip/hip_runtime.h>
#include <stdexcept>
#include "CudaManager.h"


hipError_t _cu_malloc(void** device_ptr, size_t size_byte)
{
	return gpuErrchk(hipMalloc(device_ptr, size_byte));
}
hipError_t _cu_free(void* device_ptr)
{
	return gpuErrchk(hipFree(device_ptr));
}
hipError_t _cu_syncDevice()
{
	return gpuErrchk(hipDeviceSynchronize());
}
hipError_t _cu_memset(void* device_ptr, size_t size_byte, int val)
{
	return gpuErrchk(hipMemset(device_ptr, val, size_byte));
}
hipError_t _cu_getResult(void* device_ptr, size_t size_byte, void* OutPtr)
{
	if (OutPtr == nullptr)
		throw std::runtime_error("copy with nullptr");
	return gpuErrchk(hipMemcpy(OutPtr, device_ptr, size_byte, hipMemcpyDeviceToHost));
}

hipError_t _cu_copyToDevice(void* device_ptr, size_t size_byte, const void* InPtr)
{
	if (InPtr == nullptr)
		throw std::runtime_error("copy with nullptr");
	return gpuErrchk(hipMemcpy(device_ptr, InPtr, size_byte, hipMemcpyHostToDevice));
}
hipError_t _cu_getResult(void* device_ptr, size_t size_byte, const void* OutPtr)
{
	// do nothing
	return hipSuccess;
}